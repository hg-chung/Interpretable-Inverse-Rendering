#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const int S, int W, int H,
	float focal_x, float focal_y,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ normal_opacity,
	const float* __restrict__ transMats,
	const float* __restrict__ colors,
	const float* __restrict__ features,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_depths,
	float * __restrict__ dL_dtransMat,
	float3* __restrict__ dL_dmean2D,
	float* __restrict__ dL_dnormal3D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors,
	float* __restrict__ dL_dfeatures)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = {(float)pix.x, (float)pix.y};

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_features[NUM_BASIS* 3 * BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];
	// __shared__ float collected_depths[BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];

#if RENDER_AXUTILITY
	float dL_dreg;
	float dL_ddepth;
	float dL_daccum;
	float dL_dnormal2D[3];
	const int median_contributor = inside ? n_contrib[pix_id + H * W] : 0;
	float dL_dmedian_depth;
	float dL_dmax_dweight;
	float dL_dbasis_weights[NUM_BASIS*3];

	if (inside) {
		dL_ddepth = dL_depths[DEPTH_OFFSET * H * W + pix_id];
		dL_daccum = dL_depths[ALPHA_OFFSET * H * W + pix_id];
		dL_dreg = dL_depths[DISTORTION_OFFSET * H * W + pix_id];
		for (int i = 0; i < 3; i++) 
			dL_dnormal2D[i] = dL_depths[(NORMAL_OFFSET + i) * H * W + pix_id];

		dL_dmedian_depth = dL_depths[MIDDEPTH_OFFSET * H * W + pix_id];
		for (int i=0; i<S; i++)
			dL_dbasis_weights[i] = dL_depths[(BASIS_OFFSET+i) * H * W + pix_id];
		    //dL_dmax_dweight = dL_depths[MEDIAN_WEIGHT_OFFSET * H * W + pix_id];
	}

	// for compute gradient with respect to depth and normal
	float last_depth = 0;
	float last_normal[3] = { 0 };
	float last_feature[NUM_BASIS*3] = { 0 };
	float accum_depth_rec = 0;
	float accum_alpha_rec = 0;
	float accum_normal_rec[3] = {0};
	float accum_feature_rec[NUM_BASIS*3] = { 0 };
	// for compute gradient with respect to the distortion map
	const float final_D = inside ? final_Ts[pix_id + H * W] : 0;
	const float final_D2 = inside ? final_Ts[pix_id + 2 * H * W] : 0;
	const float final_A = 1 - T_final;
	float last_dL_dT = 0;
#endif

	if (inside){
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
				// collected_depths[block.thread_rank()] = depths[coll_id];
			for (int i = 0; i < S; i++)
				collected_features[i * BLOCK_SIZE + block.thread_rank()] = features[coll_id * S + i];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// compute ray-splat intersection as before
			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];
			float3 k = pix.x * Tw - Tu;
			float3 l = pix.y * Tw - Tv;
			float3 p = cross(k, l);
			if (p.z == 0.0) continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); 

			// compute intersection and depth
			float rho = min(rho3d, rho2d);
			float c_d = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; 
			if (c_d < near_n) continue;
			float4 nor_o = collected_normal_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			float opa = nor_o.w;

			// accumulations

			float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, opa * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float w = alpha * T;
			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}
			
			for (int ch = 0; ch < S; ch++)
			{
				const float s = collected_features[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_feature_rec[ch] = last_alpha * last_feature[ch] + (1.f - last_alpha) * accum_feature_rec[ch];
				last_feature[ch] = s;

				const float dL_dchannel_feature = dL_dbasis_weights[ch];

				dL_dalpha += (s - accum_feature_rec[ch]) * dL_dchannel_feature;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dfeatures[global_id * S + ch]), dchannel_dcolor * dL_dchannel_feature);
			} 

			float dL_dz = 0.0f;
			float dL_dweight = 0;
#if RENDER_AXUTILITY
			const float m_d = far_n / (far_n - near_n) * (1 - near_n / c_d);
			const float dmd_dd = (far_n * near_n) / ((far_n - near_n) * c_d * c_d);
			if (contributor == median_contributor-1) {
				dL_dz += dL_dmedian_depth;
				// dL_dweight += dL_dmax_dweight;
			}
#if DETACH_WEIGHT 
			// if not detached weight, sometimes 
			// it will bia toward creating extragated 2D Gaussians near front
			dL_dweight += 0;
#else
			dL_dweight += (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_dreg;
#endif
			dL_dalpha += dL_dweight - last_dL_dT;
			// propagate the current weight W_{i} to next weight W_{i-1}
			last_dL_dT = dL_dweight * alpha + (1 - alpha) * last_dL_dT;
			const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_dreg;
			dL_dz += dL_dmd * dmd_dd;

			// Propagate gradients w.r.t ray-splat depths
			accum_depth_rec = last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = c_d;
			dL_dalpha += (c_d - accum_depth_rec) * dL_ddepth;
			// Propagate gradients w.r.t. color ray-splat alphas
			accum_alpha_rec = last_alpha * 1.0 + (1.f - last_alpha) * accum_alpha_rec;
			dL_dalpha += (1 - accum_alpha_rec) * dL_daccum;

			// Propagate gradients to per-Gaussian normals
			for (int ch = 0; ch < 3; ch++) {
				accum_normal_rec[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				last_normal[ch] = normal[ch];
				dL_dalpha += (normal[ch] - accum_normal_rec[ch]) * dL_dnormal2D[ch];
				atomicAdd((&dL_dnormal3D[global_id * 3 + ch]), alpha * T * dL_dnormal2D[ch]);
			}
#endif

			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = nor_o.w * dL_dalpha;
#if RENDER_AXUTILITY
			dL_dz += alpha * T * dL_ddepth; 
#endif

			if (rho3d <= rho2d) {
				// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
				const float2 dL_ds = {
					dL_dG * -G * s.x + dL_dz * Tw.x,
					dL_dG * -G * s.y + dL_dz * Tw.y
				};
				const float3 dz_dTw = {s.x, s.y, 1.0};
				const float dsx_pz = dL_ds.x / p.z;
				const float dsy_pz = dL_ds.y / p.z;
				const float3 dL_dp = {dsx_pz, dsy_pz, -(dsx_pz * s.x + dsy_pz * s.y)};
				const float3 dL_dk = cross(l, dL_dp);
				const float3 dL_dl = cross(dL_dp, k);

				const float3 dL_dTu = {-dL_dk.x, -dL_dk.y, -dL_dk.z};
				const float3 dL_dTv = {-dL_dl.x, -dL_dl.y, -dL_dl.z};
				const float3 dL_dTw = {
					pixf.x * dL_dk.x + pixf.y * dL_dl.x + dL_dz * dz_dTw.x, 
					pixf.x * dL_dk.y + pixf.y * dL_dl.y + dL_dz * dz_dTw.y, 
					pixf.x * dL_dk.z + pixf.y * dL_dl.z + dL_dz * dz_dTw.z};


				// Update gradients w.r.t. 3D covariance (3x3 matrix)
				atomicAdd(&dL_dtransMat[global_id * 9 + 0],  dL_dTu.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 1],  dL_dTu.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2],  dL_dTu.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 3],  dL_dTv.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 4],  dL_dTv.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5],  dL_dTv.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 6],  dL_dTw.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 7],  dL_dTw.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dTw.z);
			} else {
				// // Update gradients w.r.t. center of Gaussian 2D mean position
				const float dG_ddelx = -G * FilterInvSquare * d.x;
				const float dG_ddely = -G * FilterInvSquare * d.y;
				atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx); // not scaled
				atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely); // not scaled
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dz); // propagate depth loss
			}

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}


__device__ void compute_transmat_aabb(
	int idx, 
	const float* Ts_precomp,
	const float3* p_origs, 
	const glm::vec2* scales, 
	const glm::vec4* rots, 
	const float* projmatrix, 
	const float* viewmatrix, 
	const int W, const int H, 
	const float3* dL_dnormals,
	const float3* dL_dmean2Ds, 
	float* dL_dTs, 
	glm::vec3* dL_dmeans, 
	glm::vec2* dL_dscales,
	 glm::vec4* dL_drots)
{
	glm::mat3 T;
	float3 normal;
	glm::mat3x4 P;
	glm::mat3 R;
	glm::mat3 S;
	float3 p_orig;
	glm::vec4 rot;
	glm::vec2 scale;
	
	// Get transformation matrix of the Gaussian
	if (Ts_precomp != nullptr) {
		T = glm::mat3(
			Ts_precomp[idx * 9 + 0], Ts_precomp[idx * 9 + 1], Ts_precomp[idx * 9 + 2],
			Ts_precomp[idx * 9 + 3], Ts_precomp[idx * 9 + 4], Ts_precomp[idx * 9 + 5],
			Ts_precomp[idx * 9 + 6], Ts_precomp[idx * 9 + 7], Ts_precomp[idx * 9 + 8]
		);
		normal = {0.0, 0.0, 0.0};
	} else {
		p_orig = p_origs[idx];
		rot = rots[idx];
		scale = scales[idx];
		R = quat_to_rotmat(rot);
		S = scale_to_mat(scale, 1.0f);
		
		glm::mat3 L = R * S;
		glm::mat3x4 M = glm::mat3x4(
			glm::vec4(L[0], 0.0),
			glm::vec4(L[1], 0.0),
			glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
		);

		glm::mat4 world2ndc = glm::mat4(
			projmatrix[0], projmatrix[4], projmatrix[8], projmatrix[12],
			projmatrix[1], projmatrix[5], projmatrix[9], projmatrix[13],
			projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
			projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
		);

		glm::mat3x4 ndc2pix = glm::mat3x4(
			glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
			glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
			glm::vec4(0.0, 0.0, 0.0, 1.0)
		);

		P = world2ndc * ndc2pix;
		T = glm::transpose(M) * P;
		//normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);
		normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);
		//normal = transformVec4x3({L[0][2], L[1][2], L[2][2]}, viewmatrix);
	}

	// Update gradients w.r.t. transformation matrix of the Gaussian
	glm::mat3 dL_dT = glm::mat3(
		dL_dTs[idx*9+0], dL_dTs[idx*9+1], dL_dTs[idx*9+2],
		dL_dTs[idx*9+3], dL_dTs[idx*9+4], dL_dTs[idx*9+5],
		dL_dTs[idx*9+6], dL_dTs[idx*9+7], dL_dTs[idx*9+8]
	);
	float3 dL_dmean2D = dL_dmean2Ds[idx];
	if(dL_dmean2D.x != 0 || dL_dmean2D.y != 0)
	{
		const float distance = T[2].x * T[2].x + T[2].y * T[2].y - T[2].z * T[2].z;
		const float f = 1 / (distance);
		const float dpx_dT00 =  f * T[2].x;
		const float dpx_dT01 =  f * T[2].y;
		const float dpx_dT02 = -f * T[2].z;
		const float dpy_dT10 =  f * T[2].x;
		const float dpy_dT11 =  f * T[2].y;
		const float dpy_dT12 = -f * T[2].z;
		const float dpx_dT30 =  T[0].x * (f - 2 * f * f * T[2].x * T[2].x);
		const float dpx_dT31 =  T[0].y * (f - 2 * f * f * T[2].y * T[2].y);
		const float dpx_dT32 = -T[0].z * (f + 2 * f * f * T[2].z * T[2].z);
		const float dpy_dT30 =  T[1].x * (f - 2 * f * f * T[2].x * T[2].x);
		const float dpy_dT31 =  T[1].y * (f - 2 * f * f * T[2].y * T[2].y);
		const float dpy_dT32 = -T[1].z * (f + 2 * f * f * T[2].z * T[2].z);

		dL_dT[0].x += dL_dmean2D.x * dpx_dT00;
		dL_dT[0].y += dL_dmean2D.x * dpx_dT01;
		dL_dT[0].z += dL_dmean2D.x * dpx_dT02;
		dL_dT[1].x += dL_dmean2D.y * dpy_dT10;
		dL_dT[1].y += dL_dmean2D.y * dpy_dT11;
		dL_dT[1].z += dL_dmean2D.y * dpy_dT12;
		dL_dT[2].x += dL_dmean2D.x * dpx_dT30 + dL_dmean2D.y * dpy_dT30;
		dL_dT[2].y += dL_dmean2D.x * dpx_dT31 + dL_dmean2D.y * dpy_dT31;
		dL_dT[2].z += dL_dmean2D.x * dpx_dT32 + dL_dmean2D.y * dpy_dT32;

		if (Ts_precomp != nullptr) {
			dL_dTs[idx * 9 + 0] = dL_dT[0].x;
			dL_dTs[idx * 9 + 1] = dL_dT[0].y;
			dL_dTs[idx * 9 + 2] = dL_dT[0].z;
			dL_dTs[idx * 9 + 3] = dL_dT[1].x;
			dL_dTs[idx * 9 + 4] = dL_dT[1].y;
			dL_dTs[idx * 9 + 5] = dL_dT[1].z;
			dL_dTs[idx * 9 + 6] = dL_dT[2].x;
			dL_dTs[idx * 9 + 7] = dL_dT[2].y;
			dL_dTs[idx * 9 + 8] = dL_dT[2].z;
			return;
		}
	}
	
	if (Ts_precomp != nullptr) return;

	// Update gradients w.r.t. scaling, rotation, position of the Gaussian
	glm::mat3x4 dL_dM = P * glm::transpose(dL_dT);
	float3 dL_dtn = transformVec4x3Transpose(dL_dnormals[idx], viewmatrix);
#if DUAL_VISIABLE
	float3 p_view = transformPoint4x3(p_orig, viewmatrix);
	float cos = -sumf3(p_view * normal);
	float multiplier = cos > 0 ? 1: -1;
	dL_dtn = multiplier * dL_dtn;
#endif
	glm::mat3 dL_dRS = glm::mat3(
		glm::vec3(dL_dM[0]),
		glm::vec3(dL_dM[1]),
		glm::vec3(dL_dtn.x, dL_dtn.y, dL_dtn.z)
	);

	glm::mat3 dL_dR = glm::mat3(
		dL_dRS[0] * glm::vec3(scale.x),
		dL_dRS[1] * glm::vec3(scale.y),
		dL_dRS[2]);
	
	dL_drots[idx] = quat_to_rotmat_vjp(rot, dL_dR);
	dL_dscales[idx] = glm::vec2(
		(float)glm::dot(dL_dRS[0], R[0]),
		(float)glm::dot(dL_dRS[1], R[1])
	);
	dL_dmeans[idx] = glm::vec3(dL_dM[2]);
}

template<int C>
__global__ void preprocessCUDA(
	int P,
	const float3* means3D,
	const float* transMats,
	const int* radii,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, 
	const float focal_y,
	const float tan_fovx,
	const float tan_fovy,
	const glm::vec3* campos, 
	// grad input
	float* dL_dtransMats,
	const float* dL_dnormal3Ds,
	float* dL_dcolors,
	float3* dL_dmean2Ds,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	const int W = int(focal_x * tan_fovx * 2);
	const int H = int(focal_y * tan_fovy * 2);
	const float * Ts_precomp = (scales) ? nullptr : transMats;
	compute_transmat_aabb(
		idx, 
		Ts_precomp,
		means3D, scales, rotations, 
		projmatrix, viewmatrix, W, H, 
		(float3*)dL_dnormal3Ds, 
		dL_dmean2Ds,
		(dL_dtransMats), 
		dL_dmean3Ds, 
		dL_dscales, 
		dL_drots
	);

	// hack the gradient here for densitification
	float depth = transMats[idx * 9 + 8];
	dL_dmean2Ds[idx].x = dL_dtransMats[idx * 9 + 2] * depth * 0.5 * float(W); // to ndc 
	dL_dmean2Ds[idx].y = dL_dtransMats[idx * 9 + 5] * depth * 0.5 * float(H); // to ndc
}


void BACKWARD::preprocess(
	int P,
	const float3* means3D,
	const int* radii,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* transMats,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	const glm::vec3* campos, 
	float3* dL_dmean2Ds,
	const float* dL_dnormal3Ds,
	float* dL_dtransMats,
	float* dL_dcolors,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots)
{	
	preprocessCUDA<NUM_CHANNELS><< <(P + 255) / 256, 256 >> > (
		P,
		(float3*)means3D,
		transMats,
		radii,
		clamped,
		(glm::vec2*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		focal_x, 
		focal_y,
		tan_fovx,
		tan_fovy,
		campos,	
		dL_dtransMats,
		dL_dnormal3Ds,
		dL_dcolors,
		dL_dmean2Ds,
		dL_dmean3Ds,
		dL_dscales,
		dL_drots
	);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int S, int W, int H,
	float focal_x, float focal_y,
	const float* bg_color,
	const float2* means2D,
	const float4* normal_opacity,
	const float* colors,
	const float* features,
	const float* transMats,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_depths,
	float * dL_dtransMat,
	float3* dL_dmean2D,
	float* dL_dnormal3D,
	float* dL_dopacity,
	float* dL_dcolors,
	float* dL_dfeatures)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		S, W, H,
		focal_x, focal_y,
		bg_color,
		means2D,
		normal_opacity,
		transMats,
		colors,
		features,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_depths,
		dL_dtransMat,
		dL_dmean2D,
		dL_dnormal3D,
		dL_dopacity,
		dL_dcolors,
		dL_dfeatures
		);
}
